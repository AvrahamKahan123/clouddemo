
#include <hip/hip_runtime.h>
#include <iostream>

using namespace std;
void addVectors(int size, int * a, int * b, int * dest) {
    for (int i=0; i < size; ++i) {
        dest[i] = a[i] + b[i];
    }
}

int main() {
    int N = 100000000;
    int * a = new int[N]; // declared on heap so no segfault
    int * b = new int[N];
    int * sum = new int[N];

    for (int i = 0; i < N; ++i) {
        a[i] = i*i;
        b[i] = i*i;
    }
    addVectors(N, a, b, sum);
    if (sum[10] == 200) {
        return 0;
    }
    else {
        cout << "SEMANTIC ERROR" << endl;
    }
}