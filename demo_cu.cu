

#include <hip/hip_runtime.h>
#include <iostream>

#define numBlocks 512
#define numThreads 1024

using namespace std;

__global__
void addVectors(int blockSize, int totalSize, int * a, int * b, int * dest) {
    int idxBlock = blockIdx.x;
    int idx = blockIdx.x * blockSize + threadIdx.x;
    while (idx < (idxBlock+1)*blockSize && idx < totalSize) {
                 dest[idx] = a[idx] + b[idx];
                 idx+=numThreads;
             }
}

int main() {
    int N = 100000000;
    int * a = new int[N]; // declared on heap so no segfault
    int * b = new int[N];
    int * sum = new int[N];
    for (int i = 0; i < N; ++i) {
        a[i] = i*i;
        b[i] = i*i;
    }
    int *d_a, *d_b, *dest;
    hipMalloc((void**)&d_a, N*sizeof(int));
    hipMalloc((void**)&d_b, N*sizeof(int));
    hipMalloc((void**)&dest, N*sizeof(int));
    hipMemcpy(d_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(int), hipMemcpyHostToDevice);
    int blockSize = (N + numBlocks - 1)/numBlocks;
    addVectors<<<numBlocks, numThreads>>>(blockSize, N, d_a, d_b, dest);
    hipMemcpy(sum, dest, N * sizeof(int), hipMemcpyDeviceToHost);
    if (sum[10] == 200) {
        return 0;
    }
    else {
        cout << "SEMANTIC ERROR" << endl;
    }
}